#include "cuda/implementacion/util_cuda.h"

#include <hip/hip_runtime.h>
#include <sstream>

namespace clasificador_de_distribuciones
{
namespace cuda
{
namespace implementacion
{
bool checkCUDAError(const char *fase, std::string& msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        std::stringstream ss;
        ss<<"Cuda error: "<<fase<<": "<<hipGetErrorString(err)<<".\n";
        msg = ss.str();
        return true;
    }
    return false;
}
} // namespace implementacion
} // namespace secuencial
} // namespace clasificador_de_distribuciones